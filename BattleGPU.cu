#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>

#ifndef COMMON_H
#include "common.h"
#endif

const int TILE_SIZE = 16;
const int NUM_POKEMON = 2;
// GPU-compatible random number generation
int getRandom(int seed, int offset) {
    hiprandState state;
    hiprand_init(seed, offset, 0, &state);
    return hiprand(&state) % 100;
}

Pokemon battleGPU(Pokemon& pokemon1, Pokemon& pokemon2, int seed) {
    int moveIndex;
    Move selected;

    while (pokemon1.getHP() >= 1 && pokemon2.getHP() >= 1) {
        if (pokemon1.getSpe() > pokemon2.getSpe()) {
            moveIndex = getRandom(seed, 1) % pokemon1.getMovesNum();
            selected = pokemon1.getMove(moveIndex);

            if (selected.getCat() == 3) {
                statusCalc(pokemon1, pokemon2, selected);
            }
            else {
                damageCalc(pokemon1, pokemon2, selected);
            }

            if (pokemon2.getHP() < 1) break;

            moveIndex = getRandom(seed, 2) % pokemon2.getMovesNum();
            selected = pokemon2.getMove(moveIndex);

            if (selected.getCat() == 3) {
                statusCalc(pokemon2, pokemon1, selected);
            }
            else {
                damageCalc(pokemon2, pokemon1, selected);
            }
        }
        else {
            moveIndex = getRandom(seed, 3) % pokemon2.getMovesNum();
            selected = pokemon2.getMove(moveIndex);

            if (selected.getCat() == 3) {
                statusCalc(pokemon2, pokemon1, selected);
            }
            else {
                damageCalc(pokemon2, pokemon1, selected);
            }

            if (pokemon1.getHP() < 1) break;

            moveIndex = getRandom(seed, 4) % pokemon1.getMovesNum();
            selected = pokemon1.getMove(moveIndex);

            if (selected.getCat() == 3) {
                statusCalc(pokemon1, pokemon2, selected);
            }
            else {
                damageCalc(pokemon1, pokemon2, selected);
            }
        }
    }

    if (pokemon2.getHP() < 1) {
        return pokemon1;
    }
    else {
        return pokemon2;
    }
}

// Kernel to simulate many battles
__global__ void battleKernel(Pokemon* p1Array, Pokemon* p2Array, Pokemon* results, int numBattles) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numBattles) {
        results[idx] = pokeBattleGPU(p1Array[idx], p2Array[idx], idx + 1234); // Unique seed per thread
    }
}

bool pokeBattleGPU(Pokemon* pokemon1, Pokemon* pokemon2) {
    Pokemon* d_p1, *d_p2;

    // Allocate memory on the device
    hipMalloc((void**)&d_p1, sizeof(Pokemon) * NUM_POKEMON);
    hipMalloc((void**)&d_p2, sizeof(Pokemon) * NUM_POKEMON);

    // Copy input Pok�mon from host to device
    hipMemcpy(d_p1, &pokemon1, sizeof(Pokemon), hipMemcpyHostToDevice);
    hipMemcpy(d_p2, &pokemon2, sizeof(Pokemon), hipMemcpyHostToDevice);

    dim3 blockSize(TILE_SIZE);
    dim3 gridSize((size + TILE_SIZE - 1) / TILE_SIZE);

    // Launch 1 thread to simulate 1 battle
    battleKernel << <dimGrid, dimBlock >> > (d_p1, d_p2, 1);
    hipDeviceSynchronize(); // Wait for GPU to finish

    // Copy updated Pok�mon back to host
    hipMemcpy(&pokemon1, d_p1, sizeof(Pokemon), hipMemcpyDeviceToHost);
    hipMemcpy(&pokemon2, d_p2, sizeof(Pokemon), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_p1);
    hipFree(d_p2);

    // Determine winner
    return pokemon2.getHP() < 1;
}

