#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <unordered_map>
#include <random>
#include <string>
#include "Move.h"
#include "TYPES.h"
#include "Pokemon.h"
#include "PokeDex.h"
#include "AttackDex.h"
#include "PokemonData.h"
#include "Slowdown.h"
#include <vector>
#include <random>
#include <thread>
#include <chrono>

#ifndef COMMON_H
#include "common.h"
#endif

const int TILE_SIZE = 512;

__constant__ double statMultiplierGPU[13] = {
    2.0 / 8.0, 2.0 / 7.0, 2.0 / 6.0,
    2.0 / 5.0, 2.0 / 4.0, 2.0 / 3.0,
    2.0 / 2.0,
    3.0 / 2.0, 4.0 / 2.0, 5.0 / 2.0,
    6.0 / 2.0, 7.0 / 2.0, 8.0 / 2.0
};

__constant__ double accMultiplierGPU[13] = {
    3.0 / 9.0, 3.0 / 8.0, 3.0 / 7.0,
    3.0 / 6.0, 3.0 / 5.0, 3.0 / 4.0,
    3.0 / 3.0,
    4.0 / 3.0, 5.0 / 3.0, 6.0 / 3.0,
    7.0 / 3.0, 8.0 / 3.0, 9.0 / 3.0
};

//__constant__ double evaMultiplierGPU[13] = {
//    9.0 / 3.0, 8.0 / 3.0, 7.0 / 3.0,
//    6.0 / 3.0, 5.0 / 3.0, 4.0 / 3.0,
//    3.0 / 3.0,
//    3.0 / 4.0, 3.0 / 5.0, 3.0 / 6.0,
//    3.0 / 7.0, 3.0 / 8.0, 3.0 / 9.0
//};


// GPU-compatible random number generation
__device__ int getRandom(int seed, int offset) {
    hiprandState state;
    hiprand_init(seed, offset, 0, &state);
    return hiprand(&state) % 100;
}

__device__ static void typeMultiplierGPU(int damage, Move move, Pokemon defender, int* outArray) {
    double modifier = 1.0;
    int superEff = 0;
    int notVeryEff = 0;
    int noEff = 0;

    // Check if defender has a primary type
    if (defender.type1 != TYPES::NONE) { // Adjust as necessary to handle null or no type
        switch (defender.type1) {
        case TYPES::BUG:
            // Weak: Flying, Rock, Fire
            if (move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::FIRE) {
                modifier *= 2;
            }
            // Resist: Fighting, Ground, Grass
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::GRASS) {
                modifier /= 2;
            }
            break;

        case TYPES::DARK:
            //Weak: BUG, FAIRY, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::FAIRY || move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Resist: GHOST, DARK
            if (move.type == TYPES::GHOST || move.type == TYPES::DARK) {
                modifier = modifier / 2;
            }
            //Immune: Psychic
            if (move.type == TYPES::PSYCHIC) {
                modifier = 0;
            }
            break;

        case TYPES::DRAGON:
            //Weak: DRAGON, FAIRY, ICE
            if (move.type == TYPES::DRAGON || move.type == TYPES::FAIRY || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: FIRE, WATER, GRASS, ELECTRIC
            if (move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ELECTRIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ELECTRIC:
            //Weak: GROUND
            if (move.type == TYPES::GROUND) {
                modifier = modifier * 2;
            }
            //Resist: FLYING, ELECTRIC, STEEL
            if (move.type == TYPES::FLYING || move.type == TYPES::ELECTRIC || move.type == TYPES::STEEL) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FAIRY:
            //Weak: POISON, STEEL
            if (move.type == TYPES::POISON || move.type == TYPES::STEEL) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::FIGHTING) {
                modifier = modifier / 2;
            }
            //Immune: DRAGON
            if (move.type == TYPES::DRAGON) {
                modifier = 0;
            }
            break;

        case TYPES::FIGHTING:
            //Weak: FAIRY, FLYING, PSYCHIC
            if (move.type == TYPES::FAIRY || move.type == TYPES::FLYING || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, ROCK
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FIRE:
            //Weak: Ground, Rock, Water
            if (move.type == TYPES::GROUND || move.type == TYPES::ROCK || move.type == TYPES::WATER) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Steel, Fire, Grass, Ice, Fairy
            if (move.type == TYPES::BUG || move.type == TYPES::STEEL || move.type == TYPES::FIRE ||
                move.type == TYPES::GRASS || move.type == TYPES::ICE || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FLYING:
            //Weak: Electric, Ice, Rock
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::ICE || move.type == TYPES::ROCK) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Fighting, Grass
            if (move.type == TYPES::BUG || move.type == TYPES::FIGHTING || move.type == TYPES::GRASS) {
                modifier = modifier / 2;
            }
            //Immune: Ground
            if (move.type == TYPES::GROUND) {
                modifier = 0;
            }
            break;

        case TYPES::GHOST:
            //Weak: DARK, GHOST
            if (move.type == TYPES::DARK || move.type == TYPES::GHOST) {
                modifier = modifier * 2;
            }
            //Resist: BUG, POISON
            if (move.type == TYPES::BUG || move.type == TYPES::POISON) {
                modifier = modifier / 2;
            }
            //Immune: NORMAL, FIGHTING
            if (move.type == TYPES::NORMAL || move.type == TYPES::FIGHTING) {
                modifier = 0;
            }
            break;

        case TYPES::GRASS:
            //Weak: Bug, Fire, Flying, Ice, Poison
            if (move.type == TYPES::BUG || move.type == TYPES::FIRE || move.type == TYPES::FLYING ||
                move.type == TYPES::ICE || move.type == TYPES::POISON) {
                modifier = modifier * 2;
            }
            //Resist: Electric, Grass, Ground, Water
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS || move.type == TYPES::GROUND || move.type == TYPES::WATER) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::GROUND:
            //Weak: WATER, GRASS, ICE
            if (move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: POISON, ROCK
            if (move.type == TYPES::POISON || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            //Immune: ELECTRIC
            if (move.type == TYPES::ELECTRIC) {
                modifier = 0;
            }
            break;

        case TYPES::ICE:
            //Weak: FIGHTING, ROCK, STEEL, FIRE
            if (move.type == TYPES::FIGHTING || move.type == TYPES::ROCK || move.type == TYPES::STEEL || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: ICE
            if (move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::NORMAL:
            //Weak: FIGHTING
            if (move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Immune: GHOST
            if (move.type == TYPES::GHOST) {
                modifier = 0;
            }
            break;

        case TYPES::POISON:
            //Weak: Ground, Psychic
            if (move.type == TYPES::GROUND || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: Fighting, Poison, Bug, Grass, FAIRY
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG ||
                move.type == TYPES::GRASS || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::PSYCHIC:
            //Weak: BUG, DARK, GHOST
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG) {
                modifier = modifier * 2;
            }
            //Resist: FIGHTING, PSYCHIC
            if (move.type == TYPES::FIGHTING || move.type == TYPES::PSYCHIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ROCK:
            //Weak: FIGHTING, GROUND, STEEL, WATER, GRASS
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::STEEL ||
                move.type == TYPES::GRASS || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, POISON, FIRE
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::POISON || move.type == TYPES::FIRE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::STEEL:
            //Weak: GROUND, FIGHTING, FIRE
            if (move.type == TYPES::GROUND || move.type == TYPES::FIGHTING || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, ROCK, BUG, STEEL, GRASS, PSYCHIC, ICE, FAIRY, DRAGON
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::BUG ||
                move.type == TYPES::STEEL || move.type == TYPES::GRASS || move.type == TYPES::PSYCHIC || move.type == TYPES::ICE ||
                move.type == TYPES::FAIRY || move.type == TYPES::DRAGON) {
                modifier = modifier / 2;
            }
            //Immune: POISON
            if (move.type == TYPES::POISON) {
                modifier = 0;
            }
            break;

        case TYPES::WATER:
            //Weak: Electric, Grass
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: Steel, Fire, Water, Ice
            if (move.type == TYPES::STEEL || move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;
        }
    }

    if (defender.type2 != TYPES::NONE) { // Adjust as necessary to handle null or no type
        switch (defender.type2) {
        case TYPES::BUG:
            // Weak: Flying, Rock, Fire
            if (move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::FIRE) {
                modifier *= 2;
            }
            // Resist: Fighting, Ground, Grass
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::GRASS) {
                modifier /= 2;
            }
            break;

        case TYPES::DARK:
            //Weak: BUG, FAIRY, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::FAIRY || move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Resist: GHOST, DARK
            if (move.type == TYPES::GHOST || move.type == TYPES::DARK) {
                modifier = modifier / 2;
            }
            //Immune: Psychic
            if (move.type == TYPES::PSYCHIC) {
                modifier = 0;
            }
            break;

        case TYPES::DRAGON:
            //Weak: DRAGON, FAIRY, ICE
            if (move.type == TYPES::DRAGON || move.type == TYPES::FAIRY || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: FIRE, WATER, GRASS, ELECTRIC
            if (move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ELECTRIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ELECTRIC:
            //Weak: GROUND
            if (move.type == TYPES::GROUND) {
                modifier = modifier * 2;
            }
            //Resist: FLYING, ELECTRIC, STEEL
            if (move.type == TYPES::FLYING || move.type == TYPES::ELECTRIC || move.type == TYPES::STEEL) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FAIRY:
            //Weak: POISON, STEEL
            if (move.type == TYPES::POISON || move.type == TYPES::STEEL) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::FIGHTING) {
                modifier = modifier / 2;
            }
            //Immune: DRAGON
            if (move.type == TYPES::DRAGON) {
                modifier = 0;
            }
            break;

        case TYPES::FIGHTING:
            //Weak: FAIRY, FLYING, PSYCHIC
            if (move.type == TYPES::FAIRY || move.type == TYPES::FLYING || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, ROCK
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FIRE:
            //Weak: Ground, Rock, Water
            if (move.type == TYPES::GROUND || move.type == TYPES::ROCK || move.type == TYPES::WATER) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Steel, Fire, Grass, Ice, Fairy
            if (move.type == TYPES::BUG || move.type == TYPES::STEEL || move.type == TYPES::FIRE ||
                move.type == TYPES::GRASS || move.type == TYPES::ICE || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FLYING:
            //Weak: Electric, Ice, Rock
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::ICE || move.type == TYPES::ROCK) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Fighting, Grass
            if (move.type == TYPES::BUG || move.type == TYPES::FIGHTING || move.type == TYPES::GRASS) {
                modifier = modifier / 2;
            }
            //Immune: Ground
            if (move.type == TYPES::GROUND) {
                modifier = 0;
            }
            break;

        case TYPES::GHOST:
            //Weak: DARK, GHOST
            if (move.type == TYPES::DARK || move.type == TYPES::GHOST) {
                modifier = modifier * 2;
            }
            //Resist: BUG, POISON
            if (move.type == TYPES::BUG || move.type == TYPES::POISON) {
                modifier = modifier / 2;
            }
            //Immune: NORMAL, FIGHTING
            if (move.type == TYPES::NORMAL || move.type == TYPES::FIGHTING) {
                modifier = 0;
            }
            break;

        case TYPES::GRASS:
            //Weak: Bug, Fire, Flying, Ice, Poison
            if (move.type == TYPES::BUG || move.type == TYPES::FIRE || move.type == TYPES::FLYING ||
                move.type == TYPES::ICE || move.type == TYPES::POISON) {
                modifier = modifier * 2;
            }
            //Resist: Electric, Grass, Ground, Water
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS || move.type == TYPES::GROUND || move.type == TYPES::WATER) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::GROUND:
            //Weak: WATER, GRASS, ICE
            if (move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: POISON, ROCK
            if (move.type == TYPES::POISON || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            //Immune: ELECTRIC
            if (move.type == TYPES::ELECTRIC) {
                modifier = 0;
            }
            break;

        case TYPES::ICE:
            //Weak: FIGHTING, ROCK, STEEL, FIRE
            if (move.type == TYPES::FIGHTING || move.type == TYPES::ROCK || move.type == TYPES::STEEL || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: ICE
            if (move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::NORMAL:
            //Weak: FIGHTING
            if (move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Immune: GHOST
            if (move.type == TYPES::GHOST) {
                modifier = 0;
            }
            break;

        case TYPES::POISON:
            //Weak: Ground, Psychic
            if (move.type == TYPES::GROUND || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: Fighting, Poison, Bug, Grass, FAIRY
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG ||
                move.type == TYPES::GRASS || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::PSYCHIC:
            //Weak: BUG, DARK, GHOST
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG) {
                modifier = modifier * 2;
            }
            //Resist: FIGHTING, PSYCHIC
            if (move.type == TYPES::FIGHTING || move.type == TYPES::PSYCHIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ROCK:
            //Weak: FIGHTING, GROUND, STEEL, WATER, GRASS
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::STEEL ||
                move.type == TYPES::GRASS || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, POISON, FIRE
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::POISON || move.type == TYPES::FIRE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::STEEL:
            //Weak: GROUND, FIGHTING, FIRE
            if (move.type == TYPES::GROUND || move.type == TYPES::FIGHTING || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, ROCK, BUG, STEEL, GRASS, PSYCHIC, ICE, FAIRY, DRAGON
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::BUG ||
                move.type == TYPES::STEEL || move.type == TYPES::GRASS || move.type == TYPES::PSYCHIC || move.type == TYPES::ICE ||
                move.type == TYPES::FAIRY || move.type == TYPES::DRAGON) {
                modifier = modifier / 2;
            }
            //Immune: POISON
            if (move.type == TYPES::POISON) {
                modifier = 0;
            }
            break;

        case TYPES::WATER:
            //Weak: Electric, Grass
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: Steel, Fire, Water, Ice
            if (move.type == TYPES::STEEL || move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;
        }
    }

    if (modifier > 1) {
        superEff++;
    }
    else if (modifier == 0) {
        noEff++;
    }
    else if (modifier < 1) {
        notVeryEff++;
    }

    damage = (int)(damage * modifier);
    outArray[0] = damage;
    outArray[1] = superEff;
    outArray[2] = notVeryEff;
    outArray[3] = noEff;  // Update as necessary for no effect cases
}

__device__ static int targetHitGPU(Pokemon& attacker, Pokemon& defender, Move move) {
    int accMove = move.accuracy;

    int stage = static_cast<int>(defender.statEva[0]) - (attacker.statAcc[0]);
    if (stage > 6) stage = 6;
    else if (stage < -6) stage = -6;
    double stageMultiplier = accMultiplierGPU[stage];

    int r = getRandom(1234, 1) % 100 + 1;

    if (r <= (accMove * stageMultiplier)) {
        return 1; // Hit
    }
    else {
        return 0; // Miss
    }
}

//__device__ void damageCalcGPU(Pokemon& attacker, Pokemon& defender, Move move, int seed) {
//    // Variables
//    double critCalc = 0;
//    double stab = 1.0;
//    double randFact = 0;
//    double force = 0;
//    double object = 0;
//
//    // Physical or special attack calculation
//    if (!targetHitGPU(attacker, defender, move)) {
//        // do nothing
//    }
//    else {
//        if (move.category == 1) {
//            force = attacker.statAtk[1];
//            object = defender.statDef[1];
//        }
//        else if (move.category == 2) {
//            force = attacker.statSpa[1];
//            object = defender.statSpd[1];
//        }
//
//        // Critical hit calculation using random number generator
//        int randVal = getRandom(seed, 0); // You can pass a custom seed and offset
//        if (randVal < (attacker.statSpe[1] / 2)) {
//            critCalc = 1.5;
//        }
//        else {
//            critCalc = 1.0;
//        }
//
//        // Random factor calculation
//        randFact = (randVal % 16 + 85) / 100.0;
//
//        // Same Type Attack Bonus (STAB)
//        if (move.type == attacker.type1 || move.type == attacker.type2) {
//            stab = 1.5;
//        }
//
//        // Base damage calculation
//        int baseDamage = static_cast<int>((((22) * move.power * (force / object)) / (50.0 + 2)) * critCalc * randFact * stab);
//
//        // Get type multiplier
//        int dmgArray[4] = {};
//        typeMultiplierGPU(baseDamage, move, defender, dmgArray);
//        baseDamage = dmgArray[0];
//
//        // Apply damage to defender's HP
//        defender.healthPoints -= baseDamage;
//    }
//}

__device__ void statusCalcGPU(Pokemon& attacker, Pokemon& defender, Move move) {
    // {buff/debuff, atk, def, spa, spd, spe, acc, eva}
    int* effect = move.statEffect;
    double mult;
    int multStage;

    if (effect[0] == 0) { // If buffing self
        if (effect[1] != 0) { // Buff attack
            multStage = effect[1];
            attacker.statAtk[0] += multStage;
            if (attacker.statAtk[0] > 6) {
                attacker.statAtk[0] = 6;
            }
            else if (attacker.statAtk[0] < -6) {
                attacker.statAtk[0] = -6;
            }
            multStage = attacker.statAtk[0];
            mult = statMultiplierGPU[multStage];
            int newAtk = static_cast<int>(mult * attacker.statAtk[1]);
            attacker.statAtk[1] = newAtk;
            /*std::cout << attacker.getPokeName() << "'s attack rose!" << std::endl;*/
        }
        if (effect[2] != 0) { // Buff defense
            multStage = effect[2];
            attacker.statDef[0] += multStage;
            if (attacker.statDef[0] > 6) {
                attacker.statDef[0] = 6;
            }
            else if (attacker.statDef[0] < -6) {
                attacker.statDef[0] = -6;
            }
            multStage = attacker.statDef[0];
            mult = statMultiplierGPU[multStage];
            int newDef = static_cast<int>(mult * attacker.statDef[1]);
            attacker.statDef[1] = newDef;
            //std::cout << attacker.getPokeName() << "'s defense rose!" << std::endl;
        }
        if (effect[3] != 0) { // Buff special attack
            multStage = effect[3];
            attacker.statSpa[0] += multStage;
            if (attacker.statSpa[0] > 6) {
                attacker.statSpa[0] = 6;
            }
            else if (attacker.statSpa[0] < -6) {
                attacker.statSpa[0] = -6;
            }
            multStage = attacker.statSpa[0];
            mult = statMultiplierGPU[multStage];
            int newSpa = static_cast<int>(mult * attacker.statSpa[1]);
            attacker.statSpa[1] = newSpa;
            //std::cout << attacker.getPokeName() << "'s special attack rose!" << std::endl;
        }
        if (effect[4] != 0) { // Buff special defense
            multStage = effect[4];
            attacker.statSpd[0] += multStage;
            if (attacker.statSpd[0] > 6) {
                attacker.statSpd[0] = 6;
            }
            else if (attacker.statSpd[0] < -6) {
                attacker.statSpd[0] = -6;
            }
            multStage = attacker.statSpd[0];
            mult = statMultiplierGPU[multStage];
            int newSpd = static_cast<int>(mult * attacker.statSpd[1]);
            attacker.statSpd[1] = newSpd;
            //std::cout << attacker.getPokeName() << "'s special defense rose!" << std::endl;
        }
        if (effect[5] != 0) { // Buff speed
            multStage = effect[5];
            attacker.statSpe[0] += multStage;
            if (attacker.statSpe[0] > 6) {
                attacker.statSpe[0] = 6;
            }
            else if (attacker.statSpe[0] < -6) {
                attacker.statSpe[0] = -6;
            }
            multStage = attacker.statSpe[0];
            mult = statMultiplierGPU[multStage];
            int newSpe = static_cast<int>(mult * attacker.statSpe[1]);
            attacker.statSpe[1] = newSpe;
            //std::cout << attacker.getPokeName() << "'s speed rose!" << std::endl;
        }
        if (effect[6] != 0) { // Buff accuracy
            multStage = effect[6];
            attacker.statAcc[0] += multStage;
            if (attacker.statAcc[0] > 6) {
                attacker.statAcc[0] = 6;
            }
            else if (attacker.statAcc[0] < -6) {
                attacker.statAcc[0] = -6;
            }
            //std::cout << attacker.getPokeName() << "'s accuracy rose!" << std::endl;
        }
        if (effect[7] != 0) { // Buff evasion
            multStage = effect[7];
            attacker.statEva[0] += multStage;
            if (attacker.statEva[0] > 6) {
                attacker.statEva[0] = 6;
            }
            else if (attacker.statEva[0] < -6) {
                attacker.statEva[0] = -6;
            }
            //std::cout << attacker.getPokeName() << "'s evasion rose!" << std::endl;
        }
    }
    else if (effect[0] == 1) { // Debuffing opponent
        if (effect[1] != 0) { // Debuff attack
            multStage = effect[1];
            defender.statAtk[0] += multStage;
            if (defender.statAtk[0] > 6) {
                defender.statAtk[0] = 6;
            }
            else if (defender.statAtk[0] < -6) {
                defender.statAtk[0] = -6;
            }
            multStage = defender.statAtk[0];
            mult = statMultiplierGPU[multStage];
            int newAtk = static_cast<int>(mult * defender.statAtk[1]);
            defender.statAtk[1] = newAtk;
            //std::cout << defender.getPokeName() << "'s attack fell!" << std::endl;
        }
        if (effect[2] != 0) { // Debuff defense
            multStage = effect[2];
            defender.statDef[0] += multStage;
            if (defender.statDef[0] > 6) {
                defender.statDef[0] = 6;
            }
            else if (defender.statDef[0] < -6) {
                defender.statDef[0] = -6;
            }
            multStage = defender.statDef[0];
            mult = statMultiplierGPU[multStage];
            int newDef = static_cast<int>(mult * defender.statDef[1]);
            defender.statDef[1] = newDef;
            //std::cout << defender.getPokeName() << "'s defense fell!" << std::endl;
        }
        if (effect[3] != 0) { // Debuff special attack
            multStage = effect[3];
            defender.statSpa[0] += multStage;
            if (defender.statSpa[0] > 6) {
                defender.statSpa[0] = 6;
            }
            else if (defender.statSpa[0] < -6) {
                defender.statSpa[0] = -6;
            }
            multStage = defender.statSpa[0];
            mult = statMultiplierGPU[multStage];
            int newSpa = static_cast<int>(mult * defender.statSpa[1]);
            defender.statSpa[1] = newSpa;
            //std::cout << defender.getPokeName() << "'s special attack fell!" << std::endl;
        }
        if (effect[4] != 0) { // Debuff special defense
            multStage = effect[4];
            defender.statSpd[0] += multStage;
            if (defender.statSpd[0] > 6) {
                defender.statSpd[0] = 6;
            }
            else if (defender.statSpd[0] < -6) {
                defender.statSpd[0] = -6;
            }
            multStage = defender.statSpd[0];
            mult = statMultiplierGPU[multStage];
            int newSpd = static_cast<int>(mult * defender.statSpd[1]);
            defender.statSpd[1] = newSpd;
            //std::cout << defender.getPokeName() << "'s special defense fell!" << std::endl;
        }
        if (effect[5] != 0) { // Debuff speed
            multStage = effect[5];
            defender.statSpe[0] += multStage;
            if (defender.statSpe[0] > 6) {
                defender.statSpe[0] = 6;
            }
            else if (defender.statSpe[0] < -6) {
                defender.statSpe[0] = -6;
            }
            multStage = defender.statSpe[0];
            mult = statMultiplierGPU[multStage];
            int newSpe = static_cast<int>(mult * defender.statSpe[1]);
            defender.statSpe[1] = newSpe;
            //std::cout << defender.getPokeName() << "'s speed fell!" << std::endl;
        }
        if (effect[6] != 0) { // Debuff accuracy
            multStage = effect[6];
            defender.statAcc[0] += multStage;
            if (defender.statAcc[0] > 6) {
                defender.statAcc[0] = 6;
            }
            else if (defender.statAcc[0] < -6) {
                defender.statAcc[0] = -6;
            }
            //std::cout << defender.getPokeName() << "'s accuracy fell!" << std::endl;
        }
        if (effect[7] != 0) { // Debuff evasion
            multStage = effect[7];
            defender.statAcc[0] += multStage;
            if (defender.statAcc[0] > 6) {
                defender.statAcc[0] = 6;
            }
            else if (defender.statAcc[0] < -6) {
                defender.statAcc[0] = -6;
            }
            //std::cout << defender.getPokeName() << "'s evasion fell!" << std::endl;
        }
    }
}

//__device__ void battleGPU(Pokemon& pokemon1, Pokemon& pokemon2, Pokemon& winner, int seed) {
//    int moveIndex;
//    Move selected = pokemon1.moves[0]; // initial move bc I don't want to figure out the actual problem
//
//    while (pokemon1.healthPoints >= 1 && pokemon2.healthPoints >= 1) {
//        if (pokemon1.statSpe[1] > pokemon2.statSpe[1]) {
//            moveIndex = getRandom(seed, 1) % 4;
//            selected = pokemon1.moves[moveIndex];
//
//            if (selected.category == 3) {
//                statusCalcGPU(pokemon1, pokemon2, selected);
//            }
//            else {
//                damageCalcGPU(pokemon1, pokemon2, selected, seed);
//            }
//
//            if (pokemon2.healthPoints < 1) break;
//
//            moveIndex = getRandom(seed, 2) % 4;
//            selected = pokemon2.moves[moveIndex];
//
//            if (selected.category == 3) {
//                statusCalcGPU(pokemon2, pokemon1, selected);
//            }
//            else {
//                damageCalcGPU(pokemon2, pokemon1, selected, seed);
//            }
//        }
//        else {
//            moveIndex = getRandom(seed, 3) % 4;
//            selected = pokemon2.moves[moveIndex];
//
//            if (selected.category == 3) {
//                statusCalcGPU(pokemon2, pokemon1, selected);
//            }
//            else {
//                damageCalcGPU(pokemon2, pokemon1, selected, seed);
//            }
//
//            if (pokemon1.healthPoints < 1) break;
//
//            moveIndex = getRandom(seed, 4) % 4;
//            selected = pokemon1.moves[moveIndex];
//
//            if (selected.category == 3) {
//                statusCalcGPU(pokemon1, pokemon2, selected);
//            }
//            else {
//                damageCalcGPU(pokemon1, pokemon2, selected, seed);
//            }
//        }
//    }
//
//    if (pokemon2.healthPoints < 1) {
//        winner = pokemon1;
//    }
//    else {
//        winner = pokemon2;
//    }
//}

// Kernel to simulate many battles
//__global__ void battleKernel(Pokemon* p1Array, Pokemon* p2Array, Pokemon* results, int numBattles) {
//    int idx = threadIdx.x + blockIdx.x * blockDim.x;
//    if (idx < numBattles) {
//        battleGPU(p1Array[idx], p2Array[idx], results[idx], idx + 1234); // Unique seed per thread
//    }
//}

//bool pokeBattleGPU(Pokemon* pokemon1, Pokemon* pokemon2, Pokemon* results, int NUM_POKEMON) {
//    hipError_t status;
//    Pokemon* d_p1, * d_p2, * d_results;
//    // Allocate memory on the device
//    hipMalloc((void**)&d_p1, sizeof(Pokemon) * NUM_POKEMON);
//    hipMalloc((void**)&d_p2, sizeof(Pokemon) * NUM_POKEMON);
//    hipMalloc((void**)&d_results, sizeof(Pokemon) * NUM_POKEMON);
//
//    // Correctly copy full arrays from host to device
//    hipError_t err = hipMemcpy(d_p1, pokemon1, sizeof(Pokemon) * NUM_POKEMON, hipMemcpyHostToDevice);
//    if (err != hipSuccess) {
//        printf("CUDA memcpy failed: %s\n", hipGetErrorString(err));
//        return false;
//    }
//
//    hipMemcpy(d_p2, pokemon2, sizeof(Pokemon) * NUM_POKEMON, hipMemcpyHostToDevice);
//
//    //printf("name %s\n", d_p1[0].Pokename);
//
//    dim3 blockSize(TILE_SIZE);
//    dim3 gridSize((NUM_POKEMON + TILE_SIZE - 1) / TILE_SIZE);
//
//    // Launch kernel
//    battleKernel << <gridSize, blockSize >> > (d_p1, d_p2, d_results, NUM_POKEMON);
//    hipDeviceSynchronize();
//
//    status = hipGetLastError();
//    if (status != hipSuccess) {
//        std::cerr << "Kernel failed: " << hipGetErrorString(status) << std::endl;
//        hipFree(d_p1);
//        hipFree(d_p2);
//        hipFree(d_results);
//        return false;
//    }
//
//    // Copy full results array back to host
//    hipMemcpy(results, d_results, sizeof(Pokemon) * NUM_POKEMON, hipMemcpyDeviceToHost);
//
//    // Free GPU memory
//    hipFree(d_p1);
//    hipFree(d_p2);
//    hipFree(d_results);
//
//    return true;
//}
__device__ void damageCalcGPU(int attackerAtk, int defenderDef, int& defenderHP, Move move) {
    int damage = (move.power * attackerAtk) / (defenderDef + 1);
    defenderHP -= damage;
    if (defenderHP < 0) defenderHP = 0;
}

__device__ int battleGPUNew(PokemonData* p1, PokemonData* p2, int* result, int idx, int seed) {
    int p1HP = p1->healthPoints[idx];
    int p2HP = p2->healthPoints[idx];

    while (p1HP > 0 && p2HP > 0) {
        if (p1->speed[idx] > p2->speed[idx]) {
            Move m1 = p1->moves[idx][seed % 4];
            damageCalcGPU(p1->attack[idx], p2->defense[idx], p2HP, m1);
            if (p2HP <= 0) break;

            Move m2 = p2->moves[idx][(seed + 1) % 4];
            damageCalcGPU(p2->attack[idx], p1->defense[idx], p1HP, m2);
        }
        else {
            Move m2 = p2->moves[idx][(seed + 2) % 4];
            damageCalcGPU(p2->attack[idx], p1->defense[idx], p1HP, m2);
            if (p1HP <= 0) break;

            Move m1 = p1->moves[idx][(seed + 3) % 4];
            damageCalcGPU(p1->attack[idx], p2->defense[idx], p2HP, m1);
        }
    }

    // Write the result to the result array
    if (p1HP > 0) {
        result[idx] = 1;
    }
    else {
        result[idx] = 2;
    }
}

__global__ void battleKernel(PokemonData* p1, PokemonData* p2, int* result, int numBattles) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numBattles) {
        battleGPUNew(p1, p2, result, idx, idx + 1337);  // Pass pointers
    }
}

bool pokeBattleGPUNew(PokemonData* p1, PokemonData* p2, int* result, int numBattles) {
    PokemonData* d_p1;
    PokemonData* d_p2;
    int* d_result;

    hipMalloc((void**)&d_p1, sizeof(PokemonData));
    hipMalloc((void**)&d_p2, sizeof(PokemonData));
    hipMalloc((void**)&d_result, sizeof(int) * numBattles);

    // Copy host PokemonData (single structs, each holding SoA arrays)
    hipMemcpy(d_p1, p1, sizeof(PokemonData), hipMemcpyHostToDevice);
    hipMemcpy(d_p2, p2, sizeof(PokemonData), hipMemcpyHostToDevice);

    dim3 blockSize(TILE_SIZE);
    dim3 gridSize((numBattles + TILE_SIZE - 1) / TILE_SIZE);

    // Launch the kernel
    battleKernel << <gridSize, blockSize >> > (d_p1, d_p2, d_result, numBattles);

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Launch Error: " << hipGetErrorString(err) << std::endl;
        return false;
    }

    // Copy results back
    hipMemcpy(result, d_result, sizeof(int) * numBattles, hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < numBattles; ++i) {
        std::cout << "Battle " << i + 1 << " result: Winner = Pok�mon " << result[i] << std::endl;
    }

    hipFree(d_p1);
    hipFree(d_p2);
    hipFree(d_result);

    return true;
}

