#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <unordered_map>
#include <random>
#include <string>
#include "Move.h"
#include "TYPES.h"
#include "Pokemon.h"
#include "PokeDex.h"
#include "AttackDex.h"
#include "PokemonData.h"
#include "Slowdown.h"
#include <vector>
#include <random>
#include <thread>
#include <chrono>

#ifndef COMMON_H
#include "common.h"
#endif

const int TILE_SIZE = 256;

__constant__ double statMultiplierGPU[13] = {
    2.0 / 8.0, 2.0 / 7.0, 2.0 / 6.0,
    2.0 / 5.0, 2.0 / 4.0, 2.0 / 3.0,
    2.0 / 2.0,
    3.0 / 2.0, 4.0 / 2.0, 5.0 / 2.0,
    6.0 / 2.0, 7.0 / 2.0, 8.0 / 2.0
};

__constant__ double accMultiplierGPU[13] = {
    3.0 / 9.0, 3.0 / 8.0, 3.0 / 7.0,
    3.0 / 6.0, 3.0 / 5.0, 3.0 / 4.0,
    3.0 / 3.0,
    4.0 / 3.0, 5.0 / 3.0, 6.0 / 3.0,
    7.0 / 3.0, 8.0 / 3.0, 9.0 / 3.0
};

//__constant__ double evaMultiplierGPU[13] = {
//    9.0 / 3.0, 8.0 / 3.0, 7.0 / 3.0,
//    6.0 / 3.0, 5.0 / 3.0, 4.0 / 3.0,
//    3.0 / 3.0,
//    3.0 / 4.0, 3.0 / 5.0, 3.0 / 6.0,
//    3.0 / 7.0, 3.0 / 8.0, 3.0 / 9.0
//};

__device__ static void typeMultiplierGPU(int damage, Move move, TYPES defenderType1, TYPES defenderType2, int* outArray) {
    double modifier = 1.0;
    int superEff = 0;
    int notVeryEff = 0;
    int noEff = 0;

    // Check if defender has a primary type
    if (defenderType1 != TYPES::NONE) { // Adjust as necessary to handle null or no type
        switch (defenderType1) {
        case TYPES::BUG:
            // Weak: Flying, Rock, Fire
            if (move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::FIRE) {
                modifier *= 2;
            }
            // Resist: Fighting, Ground, Grass
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::GRASS) {
                modifier /= 2;
            }
            break;

        case TYPES::DARK:
            //Weak: BUG, FAIRY, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::FAIRY || move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Resist: GHOST, DARK
            if (move.type == TYPES::GHOST || move.type == TYPES::DARK) {
                modifier = modifier / 2;
            }
            //Immune: Psychic
            if (move.type == TYPES::PSYCHIC) {
                modifier = 0;
            }
            break;

        case TYPES::DRAGON:
            //Weak: DRAGON, FAIRY, ICE
            if (move.type == TYPES::DRAGON || move.type == TYPES::FAIRY || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: FIRE, WATER, GRASS, ELECTRIC
            if (move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ELECTRIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ELECTRIC:
            //Weak: GROUND
            if (move.type == TYPES::GROUND) {
                modifier = modifier * 2;
            }
            //Resist: FLYING, ELECTRIC, STEEL
            if (move.type == TYPES::FLYING || move.type == TYPES::ELECTRIC || move.type == TYPES::STEEL) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FAIRY:
            //Weak: POISON, STEEL
            if (move.type == TYPES::POISON || move.type == TYPES::STEEL) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::FIGHTING) {
                modifier = modifier / 2;
            }
            //Immune: DRAGON
            if (move.type == TYPES::DRAGON) {
                modifier = 0;
            }
            break;

        case TYPES::FIGHTING:
            //Weak: FAIRY, FLYING, PSYCHIC
            if (move.type == TYPES::FAIRY || move.type == TYPES::FLYING || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, ROCK
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FIRE:
            //Weak: Ground, Rock, Water
            if (move.type == TYPES::GROUND || move.type == TYPES::ROCK || move.type == TYPES::WATER) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Steel, Fire, Grass, Ice, Fairy
            if (move.type == TYPES::BUG || move.type == TYPES::STEEL || move.type == TYPES::FIRE ||
                move.type == TYPES::GRASS || move.type == TYPES::ICE || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FLYING:
            //Weak: Electric, Ice, Rock
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::ICE || move.type == TYPES::ROCK) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Fighting, Grass
            if (move.type == TYPES::BUG || move.type == TYPES::FIGHTING || move.type == TYPES::GRASS) {
                modifier = modifier / 2;
            }
            //Immune: Ground
            if (move.type == TYPES::GROUND) {
                modifier = 0;
            }
            break;

        case TYPES::GHOST:
            //Weak: DARK, GHOST
            if (move.type == TYPES::DARK || move.type == TYPES::GHOST) {
                modifier = modifier * 2;
            }
            //Resist: BUG, POISON
            if (move.type == TYPES::BUG || move.type == TYPES::POISON) {
                modifier = modifier / 2;
            }
            //Immune: NORMAL, FIGHTING
            if (move.type == TYPES::NORMAL || move.type == TYPES::FIGHTING) {
                modifier = 0;
            }
            break;

        case TYPES::GRASS:
            //Weak: Bug, Fire, Flying, Ice, Poison
            if (move.type == TYPES::BUG || move.type == TYPES::FIRE || move.type == TYPES::FLYING ||
                move.type == TYPES::ICE || move.type == TYPES::POISON) {
                modifier = modifier * 2;
            }
            //Resist: Electric, Grass, Ground, Water
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS || move.type == TYPES::GROUND || move.type == TYPES::WATER) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::GROUND:
            //Weak: WATER, GRASS, ICE
            if (move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: POISON, ROCK
            if (move.type == TYPES::POISON || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            //Immune: ELECTRIC
            if (move.type == TYPES::ELECTRIC) {
                modifier = 0;
            }
            break;

        case TYPES::ICE:
            //Weak: FIGHTING, ROCK, STEEL, FIRE
            if (move.type == TYPES::FIGHTING || move.type == TYPES::ROCK || move.type == TYPES::STEEL || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: ICE
            if (move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::NORMAL:
            //Weak: FIGHTING
            if (move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Immune: GHOST
            if (move.type == TYPES::GHOST) {
                modifier = 0;
            }
            break;

        case TYPES::POISON:
            //Weak: Ground, Psychic
            if (move.type == TYPES::GROUND || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: Fighting, Poison, Bug, Grass, FAIRY
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG ||
                move.type == TYPES::GRASS || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::PSYCHIC:
            //Weak: BUG, DARK, GHOST
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG) {
                modifier = modifier * 2;
            }
            //Resist: FIGHTING, PSYCHIC
            if (move.type == TYPES::FIGHTING || move.type == TYPES::PSYCHIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ROCK:
            //Weak: FIGHTING, GROUND, STEEL, WATER, GRASS
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::STEEL ||
                move.type == TYPES::GRASS || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, POISON, FIRE
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::POISON || move.type == TYPES::FIRE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::STEEL:
            //Weak: GROUND, FIGHTING, FIRE
            if (move.type == TYPES::GROUND || move.type == TYPES::FIGHTING || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, ROCK, BUG, STEEL, GRASS, PSYCHIC, ICE, FAIRY, DRAGON
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::BUG ||
                move.type == TYPES::STEEL || move.type == TYPES::GRASS || move.type == TYPES::PSYCHIC || move.type == TYPES::ICE ||
                move.type == TYPES::FAIRY || move.type == TYPES::DRAGON) {
                modifier = modifier / 2;
            }
            //Immune: POISON
            if (move.type == TYPES::POISON) {
                modifier = 0;
            }
            break;

        case TYPES::WATER:
            //Weak: Electric, Grass
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: Steel, Fire, Water, Ice
            if (move.type == TYPES::STEEL || move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;
        }
    }

    if (defenderType2 != TYPES::NONE) { // Adjust as necessary to handle null or no type
        switch (defenderType2) {
        case TYPES::BUG:
            // Weak: Flying, Rock, Fire
            if (move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::FIRE) {
                modifier *= 2;
            }
            // Resist: Fighting, Ground, Grass
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::GRASS) {
                modifier /= 2;
            }
            break;

        case TYPES::DARK:
            //Weak: BUG, FAIRY, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::FAIRY || move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Resist: GHOST, DARK
            if (move.type == TYPES::GHOST || move.type == TYPES::DARK) {
                modifier = modifier / 2;
            }
            //Immune: Psychic
            if (move.type == TYPES::PSYCHIC) {
                modifier = 0;
            }
            break;

        case TYPES::DRAGON:
            //Weak: DRAGON, FAIRY, ICE
            if (move.type == TYPES::DRAGON || move.type == TYPES::FAIRY || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: FIRE, WATER, GRASS, ELECTRIC
            if (move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ELECTRIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ELECTRIC:
            //Weak: GROUND
            if (move.type == TYPES::GROUND) {
                modifier = modifier * 2;
            }
            //Resist: FLYING, ELECTRIC, STEEL
            if (move.type == TYPES::FLYING || move.type == TYPES::ELECTRIC || move.type == TYPES::STEEL) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FAIRY:
            //Weak: POISON, STEEL
            if (move.type == TYPES::POISON || move.type == TYPES::STEEL) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::FIGHTING) {
                modifier = modifier / 2;
            }
            //Immune: DRAGON
            if (move.type == TYPES::DRAGON) {
                modifier = 0;
            }
            break;

        case TYPES::FIGHTING:
            //Weak: FAIRY, FLYING, PSYCHIC
            if (move.type == TYPES::FAIRY || move.type == TYPES::FLYING || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, ROCK
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FIRE:
            //Weak: Ground, Rock, Water
            if (move.type == TYPES::GROUND || move.type == TYPES::ROCK || move.type == TYPES::WATER) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Steel, Fire, Grass, Ice, Fairy
            if (move.type == TYPES::BUG || move.type == TYPES::STEEL || move.type == TYPES::FIRE ||
                move.type == TYPES::GRASS || move.type == TYPES::ICE || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FLYING:
            //Weak: Electric, Ice, Rock
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::ICE || move.type == TYPES::ROCK) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Fighting, Grass
            if (move.type == TYPES::BUG || move.type == TYPES::FIGHTING || move.type == TYPES::GRASS) {
                modifier = modifier / 2;
            }
            //Immune: Ground
            if (move.type == TYPES::GROUND) {
                modifier = 0;
            }
            break;

        case TYPES::GHOST:
            //Weak: DARK, GHOST
            if (move.type == TYPES::DARK || move.type == TYPES::GHOST) {
                modifier = modifier * 2;
            }
            //Resist: BUG, POISON
            if (move.type == TYPES::BUG || move.type == TYPES::POISON) {
                modifier = modifier / 2;
            }
            //Immune: NORMAL, FIGHTING
            if (move.type == TYPES::NORMAL || move.type == TYPES::FIGHTING) {
                modifier = 0;
            }
            break;

        case TYPES::GRASS:
            //Weak: Bug, Fire, Flying, Ice, Poison
            if (move.type == TYPES::BUG || move.type == TYPES::FIRE || move.type == TYPES::FLYING ||
                move.type == TYPES::ICE || move.type == TYPES::POISON) {
                modifier = modifier * 2;
            }
            //Resist: Electric, Grass, Ground, Water
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS || move.type == TYPES::GROUND || move.type == TYPES::WATER) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::GROUND:
            //Weak: WATER, GRASS, ICE
            if (move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: POISON, ROCK
            if (move.type == TYPES::POISON || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            //Immune: ELECTRIC
            if (move.type == TYPES::ELECTRIC) {
                modifier = 0;
            }
            break;

        case TYPES::ICE:
            //Weak: FIGHTING, ROCK, STEEL, FIRE
            if (move.type == TYPES::FIGHTING || move.type == TYPES::ROCK || move.type == TYPES::STEEL || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: ICE
            if (move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::NORMAL:
            //Weak: FIGHTING
            if (move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Immune: GHOST
            if (move.type == TYPES::GHOST) {
                modifier = 0;
            }
            break;

        case TYPES::POISON:
            //Weak: Ground, Psychic
            if (move.type == TYPES::GROUND || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: Fighting, Poison, Bug, Grass, FAIRY
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG ||
                move.type == TYPES::GRASS || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::PSYCHIC:
            //Weak: BUG, DARK, GHOST
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG) {
                modifier = modifier * 2;
            }
            //Resist: FIGHTING, PSYCHIC
            if (move.type == TYPES::FIGHTING || move.type == TYPES::PSYCHIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ROCK:
            //Weak: FIGHTING, GROUND, STEEL, WATER, GRASS
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::STEEL ||
                move.type == TYPES::GRASS || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, POISON, FIRE
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::POISON || move.type == TYPES::FIRE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::STEEL:
            //Weak: GROUND, FIGHTING, FIRE
            if (move.type == TYPES::GROUND || move.type == TYPES::FIGHTING || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, ROCK, BUG, STEEL, GRASS, PSYCHIC, ICE, FAIRY, DRAGON
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::BUG ||
                move.type == TYPES::STEEL || move.type == TYPES::GRASS || move.type == TYPES::PSYCHIC || move.type == TYPES::ICE ||
                move.type == TYPES::FAIRY || move.type == TYPES::DRAGON) {
                modifier = modifier / 2;
            }
            //Immune: POISON
            if (move.type == TYPES::POISON) {
                modifier = 0;
            }
            break;

        case TYPES::WATER:
            //Weak: Electric, Grass
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: Steel, Fire, Water, Ice
            if (move.type == TYPES::STEEL || move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;
        }
    }

    if (modifier > 1) {
        superEff++;
    }
    else if (modifier == 0) {
        noEff++;
    }
    else if (modifier < 1) {
        notVeryEff++;
    }

    damage = (int)(damage * modifier);
    outArray[0] = damage;
    outArray[1] = superEff;
    outArray[2] = notVeryEff;
    outArray[3] = noEff;  // Update as necessary for no effect cases
}

// GPU-compatible random number generation
__device__ void initRandom(hiprandState* state, int seed, int idx) {
    hiprand_init(seed, idx, 0, state);
}

__device__ int getRandom(hiprandState* state) {
    return hiprand(state) % 100;
}

__device__ static int targetHitGPU(int attackerAccST, int defenderEvaST, Move move, hiprandState* rng) {
    int accMove = move.accuracy;

    int stage = static_cast<int>(attackerAccST) - (defenderEvaST);
    if (stage > 6) stage = 6;
    else if (stage < -6) stage = -6;
    double stageMultiplier = accMultiplierGPU[stage];

    int r = hiprand(rng) % 100 + 1;

    if (r <= (accMove * stageMultiplier)) {
        return 1; // Hit
    }
    else {
        return 0; // Miss
    }
}

__device__ void damageCalcGPU(
    int attackerAtk, int defenderDef,
    int attackerSpa, int defenderSpd,
    int attackerSpe,
    TYPES attackerType1, TYPES attackerType2,
    TYPES defenderType1, TYPES defenderType2,
    int& defenderHP,
    Move move,
    int attackerAccST, int defenderEvaST,
    hiprandState* rng
) {
    double critMult = 1.0;
    double stab = 1.0;
    double randFact = 1.0;
    double force = 0.0;
    double object = 0.0;

    if (!targetHitGPU(attackerAccST, defenderEvaST, move, rng)) {
        return; // Move missed
    }

    // Choose attacking and defending stats based on move category
    if (move.category == 1) { // Physical
        force = attackerAtk;
        object = defenderDef;
    }
    else if (move.category == 2) { // Special
        force = attackerSpa;
        object = defenderSpd;
    }

    // Critical hit calculation
    int randVal = hiprand(rng) % 256;
    if (randVal < (attackerSpe / 2)) {
        critMult = 1.5;
    }

    // Random damage variance (85% - 100%)
    randFact = (randVal % 16 + 85) / 100.0;

    // Same Type Attack Bonus
    if (move.type == attackerType1 || move.type == attackerType2) {
        stab = 1.5;
    }

    // Base damage formula
    int baseDamage = static_cast<int>(
        (((22.0 * move.power * (force / object)) / (50.0 + 2))
            * critMult * randFact * stab)
        );

    // Apply type effectiveness
    int dmgArray[4] = {};
    typeMultiplierGPU(baseDamage, move, defenderType1, defenderType2, dmgArray);
    baseDamage = dmgArray[0];
    if (baseDamage < 1) baseDamage = 1;

    // Inflict damage
    defenderHP -= baseDamage;
}

__device__ void statusCalcGPU(
    int attackerAtkST, int defenderAtkST,
    int attackerDefST, int defenderDefST,
    int attackerSpaST, int defenderSpaST,
    int attackerSpdST, int defenderSpdST,
    int attackerSpeST, int defenderSpeST,
    int attackerAccST, int defenderAccST,
    int attackerEvaST, int defenderEvaST,
    int attackerAtk, int defenderAtk,
    int attackerDef, int defenderDef,
    int attackerSpa, int defenderSpa,
    int attackerSpd, int defenderSpd,
    int attackerSpe, int defenderSpe, Move move
) {
    // {buff/debuff, atk, def, spa, spd, spe, acc, eva}
    int* effect = move.statEffect;
    double mult;
    int multStage;

    if (effect[0] == 0) { // If buffing self
        if (effect[1] != 0) { // Buff attack
            multStage = effect[1];
            attackerAtkST += multStage;
            if (attackerAtkST > 6) {
                attackerAtkST = 6;
            }
            else if (attackerAtkST < -6) {
                attackerAtkST = -6;
            }
            multStage = attackerAtkST;
            mult = statMultiplierGPU[multStage];
            int newAtk = static_cast<int>(mult * attackerAtk);
            attackerAtk = newAtk;
            /*std::cout << attacker.getPokeName() << "'s attack rose!" << std::endl;*/
        }
        if (effect[2] != 0) { // Buff defense
            multStage = effect[2];
            attackerDefST += multStage;
            if (attackerDefST > 6) {
                attackerDefST = 6;
            }
            else if (attackerDefST < -6) {
                attackerDefST = -6;
            }
            multStage = attackerDefST;
            mult = statMultiplierGPU[multStage];
            int newDef = static_cast<int>(mult * attackerDef);
            attackerDef = newDef;
            //std::cout << attacker.getPokeName() << "'s defense rose!" << std::endl;
        }
        if (effect[3] != 0) { // Buff special attack
            multStage = effect[3];
            attackerSpaST += multStage;
            if (attackerSpaST > 6) {
                attackerSpaST = 6;
            }
            else if (attackerSpaST < -6) {
                attackerSpaST = -6;
            }
            multStage = attackerSpaST;
            mult = statMultiplierGPU[multStage];
            int newSpa = static_cast<int>(mult * attackerSpa);
            attackerSpa = newSpa;
            //std::cout << attacker.getPokeName() << "'s special attack rose!" << std::endl;
        }
        if (effect[4] != 0) { // Buff special defense
            multStage = effect[4];
            attackerSpdST += multStage;
            if (attackerSpdST > 6) {
                attackerSpdST = 6;
            }
            else if (attackerSpdST < -6) {
                attackerSpdST = -6;
            }
            multStage = attackerSpdST;
            mult = statMultiplierGPU[multStage];
            int newSpd = static_cast<int>(mult * attackerSpd);
            attackerSpd = newSpd;
            //std::cout << attacker.getPokeName() << "'s special defense rose!" << std::endl;
        }
        if (effect[5] != 0) { // Buff speed
            multStage = effect[5];
            attackerSpeST += multStage;
            if (attackerSpeST > 6) {
                attackerSpeST = 6;
            }
            else if (attackerSpeST < -6) {
                attackerSpeST = -6;
            }
            multStage = attackerSpeST;
            mult = statMultiplierGPU[multStage];
            int newSpe = static_cast<int>(mult * attackerSpeST);
            attackerSpeST = newSpe;
            //std::cout << attacker.getPokeName() << "'s speed rose!" << std::endl;
        }
        if (effect[6] != 0) { // Buff accuracy
            multStage = effect[6];
            attackerAccST += multStage;
            if (attackerAccST > 6) {
                attackerAccST = 6;
            }
            else if (attackerAccST < -6) {
                attackerAccST = -6;
            }
            //std::cout << attacker.getPokeName() << "'s accuracy rose!" << std::endl;
        }
        if (effect[7] != 0) { // Buff evasion
            multStage = effect[7];
            attackerEvaST += multStage;
            if (attackerEvaST > 6) {
                attackerEvaST = 6;
            }
            else if (attackerEvaST < -6) {
                attackerEvaST = -6;
            }
            //std::cout << attacker.getPokeName() << "'s evasion rose!" << std::endl;
        }
    }
    else if (effect[0] == 1) { // Debuffing opponent
        if (effect[1] != 0) { // Debuff attack
            multStage = effect[1];
            defenderAtkST += multStage;
            if (defenderAtkST > 6) {
                defenderAtkST = 6;
            }
            else if (defenderAtkST < -6) {
                defenderAtkST = -6;
            }
            multStage = defenderAtkST;
            mult = statMultiplierGPU[multStage];
            int newAtk = static_cast<int>(mult * defenderAtk);
            defenderAtk = newAtk;
            //std::cout << defender.getPokeName() << "'s attack fell!" << std::endl;
        }
        if (effect[2] != 0) { // Debuff defense
            multStage = effect[2];
            defenderDefST += multStage;
            if (defenderDefST > 6) {
                defenderDefST = 6;
            }
            else if (defenderDefST < -6) {
                defenderDefST = -6;
            }
            multStage = defenderDefST;
            mult = statMultiplierGPU[multStage];
            int newDef = static_cast<int>(mult * defenderDef);
            defenderDef = newDef;
            //std::cout << defender.getPokeName() << "'s defense fell!" << std::endl;
        }
        if (effect[3] != 0) { // Debuff special attack
            multStage = effect[3];
            defenderSpaST += multStage;
            if (defenderSpaST > 6) {
                defenderSpaST = 6;
            }
            else if (defenderSpaST < -6) {
                defenderSpaST = -6;
            }
            multStage = defenderSpaST;
            mult = statMultiplierGPU[multStage];
            int newSpa = static_cast<int>(mult * defenderSpa);
            defenderSpa = newSpa;
            //std::cout << defender.getPokeName() << "'s special attack fell!" << std::endl;
        }
        if (effect[4] != 0) { // Debuff special defense
            multStage = effect[4];
            defenderSpdST += multStage;
            if (defenderSpdST > 6) {
                defenderSpdST = 6;
            }
            else if (defenderSpdST < -6) {
                defenderSpdST = -6;
            }
            multStage = defenderSpdST;
            mult = statMultiplierGPU[multStage];
            int newSpd = static_cast<int>(mult * defenderSpd);
            defenderSpd = newSpd;
            //std::cout << defender.getPokeName() << "'s special defense fell!" << std::endl;
        }
        if (effect[5] != 0) { // Debuff speed
            multStage = effect[5];
            defenderSpeST += multStage;
            if (defenderSpeST > 6) {
                defenderSpeST = 6;
            }
            else if (defenderSpeST < -6) {
                defenderSpeST = -6;
            }
            multStage = defenderSpeST;
            mult = statMultiplierGPU[multStage];
            int newSpe = static_cast<int>(mult * defenderSpeST);
            defenderSpeST = newSpe;
            //std::cout << defender.getPokeName() << "'s speed fell!" << std::endl;
        }
        if (effect[6] != 0) { // Debuff accuracy
            multStage = effect[6];
            defenderAccST += multStage;
            if (defenderAccST > 6) {
                defenderAccST = 6;
            }
            else if (defenderAccST < -6) {
                defenderAccST = -6;
            }
            //std::cout << defender.getPokeName() << "'s accuracy fell!" << std::endl;
        }
        if (effect[7] != 0) { // Debuff evasion
            multStage = effect[7];
            defenderEvaST += multStage;
            if (defenderEvaST > 6) {
                defenderEvaST = 6;
            }
            else if (defenderEvaST < -6) {
                defenderEvaST = -6;
            }
            //std::cout << defender.getPokeName() << "'s evasion fell!" << std::endl;
        }
    }
}

__device__ void battleGPUNew(PokemonData* p1, PokemonData* p2, int* result, int idx, hiprandState* rng) {
    int canUse = 0;
    while (p1->healthPoints[idx] > 0 && p2->healthPoints[idx] > 0) {
        Move selected = p1->moves[0][0]; //placeholder for initialization
        if (p1->speed[idx] > p2->speed[idx]) {
            while (!canUse) {
                selected = p1->moves[idx][hiprand(rng) % 4];
                if (selected.pp < 1) {
                    //do nothing i.e. try for another move
                }
                else {
                    selected.pp -=1;
                    canUse++;
                }
            }
            canUse--;

            if (selected.category == 3) {
                statusCalcGPU(
                    p1->statAtk[idx][0], p2->statAtk[idx][0],
                    p1->statDef[idx][0], p2->statDef[idx][0],
                    p1->statSpa[idx][0], p2->statSpa[idx][0],
                    p1->statSpd[idx][0], p2->statSpd[idx][0],
                    p1->statSpe[idx][0], p2->statSpe[idx][0],
                    p1->statAcc[idx][0], p2->statAcc[idx][0],
                    p1->statEva[idx][0], p2->statEva[idx][0],
                    p1->statAtk[idx][1], p2->statAtk[idx][1],
                    p1->statDef[idx][1], p2->statDef[idx][1],
                    p1->statSpa[idx][1], p2->statSpa[idx][1],
                    p1->statSpd[idx][1], p2->statSpd[idx][1],
                    p1->statSpe[idx][1], p2->statSpe[idx][1],
                    selected
                );
            }
            else {
                damageCalcGPU(
                    p1->attack[idx], p2->defense[idx],
                    p1->specialAttack[idx], p2->specialDefense[idx],
                    p1->speed[idx],
                    p1->type1[idx], p1->type2[idx],
                    p2->type1[idx], p2->type2[idx],
                    p2->healthPoints[idx],
                    selected,
                    p1->statAcc[idx][0], p2->statEva[idx][0],
                    rng
                );
            }

            if (p2->healthPoints[idx] < 1) break;

            while (!canUse) {
                selected = p2->moves[idx][hiprand(rng) % 4];
                if (selected.pp < 1) {
                    //do nothing i.e. try for another move
                }
                else {
                    selected.pp -=1;
                    canUse++;
                }
            }
            canUse--;

            if (selected.category == 3) {
                statusCalcGPU(
                    p2->statAtk[idx][0], p1->statAtk[idx][0],
                    p2->statDef[idx][0], p1->statDef[idx][0],
                    p2->statSpa[idx][0], p1->statSpa[idx][0],
                    p2->statSpd[idx][0], p1->statSpd[idx][0],
                    p2->statSpe[idx][0], p1->statSpe[idx][0],
                    p2->statAcc[idx][0], p1->statAcc[idx][0],
                    p2->statEva[idx][0], p1->statEva[idx][0],
                    p2->statAtk[idx][1], p1->statAtk[idx][1],
                    p2->statDef[idx][1], p1->statDef[idx][1],
                    p2->statSpa[idx][1], p1->statSpa[idx][1],
                    p2->statSpd[idx][1], p1->statSpd[idx][1],
                    p2->statSpe[idx][1], p1->statSpe[idx][1],
                    selected
                );
            }
            else {
                damageCalcGPU(
                    p2->attack[idx], p1->defense[idx],
                    p2->specialAttack[idx], p1->specialDefense[idx],
                    p2->speed[idx],
                    p2->type1[idx], p2->type2[idx],
                    p1->type1[idx], p1->type2[idx],
                    p1->healthPoints[idx],
                    selected,
                    p2->statAcc[idx][0], p1->statEva[idx][0],
                    rng
                );
            }

        }
        else {
            while (!canUse) {
                selected = p2->moves[idx][hiprand(rng) % 4];
                if (selected.pp < 1) {
                    //do nothing i.e. try for another move
                }
                else {
                    selected.pp -=1;
                    canUse++;
                }
            }
            canUse--;

            if (selected.category == 3) {
                statusCalcGPU(
                    p2->statAtk[idx][0], p1->statAtk[idx][0],
                    p2->statDef[idx][0], p1->statDef[idx][0],
                    p2->statSpa[idx][0], p1->statSpa[idx][0],
                    p2->statSpd[idx][0], p1->statSpd[idx][0],
                    p2->statSpe[idx][0], p1->statSpe[idx][0],
                    p2->statAcc[idx][0], p1->statAcc[idx][0],
                    p2->statEva[idx][0], p1->statEva[idx][0],
                    p2->statAtk[idx][1], p1->statAtk[idx][1],
                    p2->statDef[idx][1], p1->statDef[idx][1],
                    p2->statSpa[idx][1], p1->statSpa[idx][1],
                    p2->statSpd[idx][1], p1->statSpd[idx][1],
                    p2->statSpe[idx][1], p1->statSpe[idx][1],
                    selected
                );
            }
            else {
                damageCalcGPU(
                    p2->attack[idx], p1->defense[idx],
                    p2->specialAttack[idx], p1->specialDefense[idx],
                    p2->speed[idx],
                    p2->type1[idx], p2->type2[idx],
                    p1->type1[idx], p1->type2[idx],
                    p1->healthPoints[idx],
                    selected,
                    p2->statAcc[idx][0], p1->statEva[idx][0],
                    rng
                );
            }

            if (p1->healthPoints[idx] < 1) break;

            while (!canUse) {
                selected = p1->moves[idx][hiprand(rng) % 4];
                if (selected.pp < 1) {
                    //do nothing i.e. try for another move
                }
                else {
                    selected.pp -=1;
                    canUse++;
                }
            }
            canUse--;

            if (selected.category == 3) {
                statusCalcGPU(
                    p1->statAtk[idx][0], p2->statAtk[idx][0],
                    p1->statDef[idx][0], p2->statDef[idx][0],
                    p1->statSpa[idx][0], p2->statSpa[idx][0],
                    p1->statSpd[idx][0], p2->statSpd[idx][0],
                    p1->statSpe[idx][0], p2->statSpe[idx][0],
                    p1->statAcc[idx][0], p2->statAcc[idx][0],
                    p1->statEva[idx][0], p2->statEva[idx][0],
                    p1->statAtk[idx][1], p2->statAtk[idx][1],
                    p1->statDef[idx][1], p2->statDef[idx][1],
                    p1->statSpa[idx][1], p2->statSpa[idx][1],
                    p1->statSpd[idx][1], p2->statSpd[idx][1],
                    p1->statSpe[idx][1], p2->statSpe[idx][1],
                    selected
                );
            }
            else {
                damageCalcGPU(
                    p1->attack[idx], p2->defense[idx],
                    p1->specialAttack[idx], p2->specialDefense[idx],
                    p1->speed[idx],
                    p1->type1[idx], p1->type2[idx],
                    p2->type1[idx], p2->type2[idx],
                    p2->healthPoints[idx],
                    selected,
                    p1->statAcc[idx][0], p2->statEva[idx][0],
                    rng
                );
            }

        }
    }

    // Write the result to the result array
    if (p2->healthPoints[idx] < 1) {
        result[idx] = 1;
    }
    else {
        result[idx] = 2;
    }
}

__global__ void battleKernel(PokemonData* p1, PokemonData* p2, int* result, int numBattles) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numBattles) {
        hiprandState rng;
        hiprand_init(12345, idx, 0, &rng);
        battleGPUNew(p1, p2, result, idx, &rng);  // Pass pointers
    }
}

bool pokeBattleGPUNew(PokemonData* p1, PokemonData* p2, int* result, int numBattles) {
    PokemonData* d_p1;
    PokemonData* d_p2;
    int* d_result;

    hipMalloc((void**)&d_p1, sizeof(PokemonData));
    hipMalloc((void**)&d_p2, sizeof(PokemonData));
    hipMalloc((void**)&d_result, sizeof(int) * numBattles);

    // Copy host PokemonData (single structs, each holding SoA arrays)
    hipMemcpy(d_p1, p1, sizeof(PokemonData), hipMemcpyHostToDevice);
    hipMemcpy(d_p2, p2, sizeof(PokemonData), hipMemcpyHostToDevice);

    dim3 blockSize(TILE_SIZE);
    dim3 gridSize((numBattles + TILE_SIZE - 1) / TILE_SIZE);

    // Launch the kernel
    battleKernel << <gridSize, blockSize >> > (d_p1, d_p2, d_result, numBattles);

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Launch Error: " << hipGetErrorString(err) << std::endl;
        return false;
    }

    // Copy results back
    hipMemcpy(result, d_result, sizeof(int) * numBattles, hipMemcpyDeviceToHost);

    // Print results

    hipFree(d_p1);
    hipFree(d_p2);
    hipFree(d_result);

    return true;
}

