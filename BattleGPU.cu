#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

#ifndef COMMON_H
#include "common.h"
#endif

#include <iostream>
#include <unordered_map>
#include <random>
#include <string>
#include "Move.h"
#include "TYPES.h"
#include "Pokemon.h"
#include "PokeDex.h"
#include "AttackDex.h"
#include "Slowdown.h"
#include <vector>
#include <random>
#include <thread>
#include <chrono>

const int TILE_SIZE = 16;
__device__ static int dmgArray[4];

__constant__ double statMultiplierGPU[13] = {
    2.0 / 8.0, 2.0 / 7.0, 2.0 / 6.0,
    2.0 / 5.0, 2.0 / 4.0, 2.0 / 3.0,
    2.0 / 2.0,
    3.0 / 2.0, 4.0 / 2.0, 5.0 / 2.0,
    6.0 / 2.0, 7.0 / 2.0, 8.0 / 2.0
};

__constant__ double accMultiplierGPU[13] = {
    3.0 / 9.0, 3.0 / 8.0, 3.0 / 7.0,
    3.0 / 6.0, 3.0 / 5.0, 3.0 / 4.0,
    3.0 / 3.0,
    4.0 / 3.0, 5.0 / 3.0, 6.0 / 3.0,
    7.0 / 3.0, 8.0 / 3.0, 9.0 / 3.0
};

__constant__ double evaMultiplierGPU[13] = {
    9.0 / 3.0, 8.0 / 3.0, 7.0 / 3.0,
    6.0 / 3.0, 5.0 / 3.0, 4.0 / 3.0,
    3.0 / 3.0,
    3.0 / 4.0, 3.0 / 5.0, 3.0 / 6.0,
    3.0 / 7.0, 3.0 / 8.0, 3.0 / 9.0
};


// GPU-compatible random number generation
__device__ int getRandom(int seed, int offset) {
    hiprandState state;
    hiprand_init(seed, offset, 0, &state);
    return hiprand(&state) % 100;
}

__device__ static void typeMultiplierGPU(int damage, Move move, Pokemon defender) {
    double modifier = 1.0;
    int superEff = 0;
    int notVeryEff = 0;
    int noEff = 0;

    // Check if defender has a primary type
    if (defender.type1 != TYPES::NONE) { // Adjust as necessary to handle null or no type
        switch (defender.type1) {
        case TYPES::BUG:
            // Weak: Flying, Rock, Fire
            if (move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::FIRE) {
                modifier *= 2;
            }
            // Resist: Fighting, Ground, Grass
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::GRASS) {
                modifier /= 2;
            }
            break;

        case TYPES::DARK:
            //Weak: BUG, FAIRY, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::FAIRY || move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Resist: GHOST, DARK
            if (move.type == TYPES::GHOST || move.type == TYPES::DARK) {
                modifier = modifier / 2;
            }
            //Immune: Psychic
            if (move.type == TYPES::PSYCHIC) {
                modifier = 0;
            }
            break;

        case TYPES::DRAGON:
            //Weak: DRAGON, FAIRY, ICE
            if (move.type == TYPES::DRAGON || move.type == TYPES::FAIRY || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: FIRE, WATER, GRASS, ELECTRIC
            if (move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ELECTRIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ELECTRIC:
            //Weak: GROUND
            if (move.type == TYPES::GROUND) {
                modifier = modifier * 2;
            }
            //Resist: FLYING, ELECTRIC, STEEL
            if (move.type == TYPES::FLYING || move.type == TYPES::ELECTRIC || move.type == TYPES::STEEL) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FAIRY:
            //Weak: POISON, STEEL
            if (move.type == TYPES::POISON || move.type == TYPES::STEEL) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::FIGHTING) {
                modifier = modifier / 2;
            }
            //Immune: DRAGON
            if (move.type == TYPES::DRAGON) {
                modifier = 0;
            }
            break;

        case TYPES::FIGHTING:
            //Weak: FAIRY, FLYING, PSYCHIC
            if (move.type == TYPES::FAIRY || move.type == TYPES::FLYING || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, ROCK
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FIRE:
            //Weak: Ground, Rock, Water
            if (move.type == TYPES::GROUND || move.type == TYPES::ROCK || move.type == TYPES::WATER) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Steel, Fire, Grass, Ice, Fairy
            if (move.type == TYPES::BUG || move.type == TYPES::STEEL || move.type == TYPES::FIRE ||
                move.type == TYPES::GRASS || move.type == TYPES::ICE || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FLYING:
            //Weak: Electric, Ice, Rock
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::ICE || move.type == TYPES::ROCK) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Fighting, Grass
            if (move.type == TYPES::BUG || move.type == TYPES::FIGHTING || move.type == TYPES::GRASS) {
                modifier = modifier / 2;
            }
            //Immune: Ground
            if (move.type == TYPES::GROUND) {
                modifier = 0;
            }
            break;

        case TYPES::GHOST:
            //Weak: DARK, GHOST
            if (move.type == TYPES::DARK || move.type == TYPES::GHOST) {
                modifier = modifier * 2;
            }
            //Resist: BUG, POISON
            if (move.type == TYPES::BUG || move.type == TYPES::POISON) {
                modifier = modifier / 2;
            }
            //Immune: NORMAL, FIGHTING
            if (move.type == TYPES::NORMAL || move.type == TYPES::FIGHTING) {
                modifier = 0;
            }
            break;

        case TYPES::GRASS:
            //Weak: Bug, Fire, Flying, Ice, Poison
            if (move.type == TYPES::BUG || move.type == TYPES::FIRE || move.type == TYPES::FLYING ||
                move.type == TYPES::ICE || move.type == TYPES::POISON) {
                modifier = modifier * 2;
            }
            //Resist: Electric, Grass, Ground, Water
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS || move.type == TYPES::GROUND || move.type == TYPES::WATER) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::GROUND:
            //Weak: WATER, GRASS, ICE
            if (move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: POISON, ROCK
            if (move.type == TYPES::POISON || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            //Immune: ELECTRIC
            if (move.type == TYPES::ELECTRIC) {
                modifier = 0;
            }
            break;

        case TYPES::ICE:
            //Weak: FIGHTING, ROCK, STEEL, FIRE
            if (move.type == TYPES::FIGHTING || move.type == TYPES::ROCK || move.type == TYPES::STEEL || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: ICE
            if (move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::NORMAL:
            //Weak: FIGHTING
            if (move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Immune: GHOST
            if (move.type == TYPES::GHOST) {
                modifier = 0;
            }
            break;

        case TYPES::POISON:
            //Weak: Ground, Psychic
            if (move.type == TYPES::GROUND || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: Fighting, Poison, Bug, Grass, FAIRY
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG ||
                move.type == TYPES::GRASS || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::PSYCHIC:
            //Weak: BUG, DARK, GHOST
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG) {
                modifier = modifier * 2;
            }
            //Resist: FIGHTING, PSYCHIC
            if (move.type == TYPES::FIGHTING || move.type == TYPES::PSYCHIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ROCK:
            //Weak: FIGHTING, GROUND, STEEL, WATER, GRASS
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::STEEL ||
                move.type == TYPES::GRASS || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, POISON, FIRE
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::POISON || move.type == TYPES::FIRE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::STEEL:
            //Weak: GROUND, FIGHTING, FIRE
            if (move.type == TYPES::GROUND || move.type == TYPES::FIGHTING || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, ROCK, BUG, STEEL, GRASS, PSYCHIC, ICE, FAIRY, DRAGON
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::BUG ||
                move.type == TYPES::STEEL || move.type == TYPES::GRASS || move.type == TYPES::PSYCHIC || move.type == TYPES::ICE ||
                move.type == TYPES::FAIRY || move.type == TYPES::DRAGON) {
                modifier = modifier / 2;
            }
            //Immune: POISON
            if (move.type == TYPES::POISON) {
                modifier = 0;
            }
            break;

        case TYPES::WATER:
            //Weak: Electric, Grass
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: Steel, Fire, Water, Ice
            if (move.type == TYPES::STEEL || move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;
        }
    }

    if (defender.type2 != TYPES::NONE) { // Adjust as necessary to handle null or no type
        switch (defender.type2) {
        case TYPES::BUG:
            // Weak: Flying, Rock, Fire
            if (move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::FIRE) {
                modifier *= 2;
            }
            // Resist: Fighting, Ground, Grass
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::GRASS) {
                modifier /= 2;
            }
            break;

        case TYPES::DARK:
            //Weak: BUG, FAIRY, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::FAIRY || move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Resist: GHOST, DARK
            if (move.type == TYPES::GHOST || move.type == TYPES::DARK) {
                modifier = modifier / 2;
            }
            //Immune: Psychic
            if (move.type == TYPES::PSYCHIC) {
                modifier = 0;
            }
            break;

        case TYPES::DRAGON:
            //Weak: DRAGON, FAIRY, ICE
            if (move.type == TYPES::DRAGON || move.type == TYPES::FAIRY || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: FIRE, WATER, GRASS, ELECTRIC
            if (move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ELECTRIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ELECTRIC:
            //Weak: GROUND
            if (move.type == TYPES::GROUND) {
                modifier = modifier * 2;
            }
            //Resist: FLYING, ELECTRIC, STEEL
            if (move.type == TYPES::FLYING || move.type == TYPES::ELECTRIC || move.type == TYPES::STEEL) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FAIRY:
            //Weak: POISON, STEEL
            if (move.type == TYPES::POISON || move.type == TYPES::STEEL) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, FIGHTING
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::FIGHTING) {
                modifier = modifier / 2;
            }
            //Immune: DRAGON
            if (move.type == TYPES::DRAGON) {
                modifier = 0;
            }
            break;

        case TYPES::FIGHTING:
            //Weak: FAIRY, FLYING, PSYCHIC
            if (move.type == TYPES::FAIRY || move.type == TYPES::FLYING || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: BUG, DARK, ROCK
            if (move.type == TYPES::BUG || move.type == TYPES::DARK || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FIRE:
            //Weak: Ground, Rock, Water
            if (move.type == TYPES::GROUND || move.type == TYPES::ROCK || move.type == TYPES::WATER) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Steel, Fire, Grass, Ice, Fairy
            if (move.type == TYPES::BUG || move.type == TYPES::STEEL || move.type == TYPES::FIRE ||
                move.type == TYPES::GRASS || move.type == TYPES::ICE || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::FLYING:
            //Weak: Electric, Ice, Rock
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::ICE || move.type == TYPES::ROCK) {
                modifier = modifier * 2;
            }
            //Resist: Bug, Fighting, Grass
            if (move.type == TYPES::BUG || move.type == TYPES::FIGHTING || move.type == TYPES::GRASS) {
                modifier = modifier / 2;
            }
            //Immune: Ground
            if (move.type == TYPES::GROUND) {
                modifier = 0;
            }
            break;

        case TYPES::GHOST:
            //Weak: DARK, GHOST
            if (move.type == TYPES::DARK || move.type == TYPES::GHOST) {
                modifier = modifier * 2;
            }
            //Resist: BUG, POISON
            if (move.type == TYPES::BUG || move.type == TYPES::POISON) {
                modifier = modifier / 2;
            }
            //Immune: NORMAL, FIGHTING
            if (move.type == TYPES::NORMAL || move.type == TYPES::FIGHTING) {
                modifier = 0;
            }
            break;

        case TYPES::GRASS:
            //Weak: Bug, Fire, Flying, Ice, Poison
            if (move.type == TYPES::BUG || move.type == TYPES::FIRE || move.type == TYPES::FLYING ||
                move.type == TYPES::ICE || move.type == TYPES::POISON) {
                modifier = modifier * 2;
            }
            //Resist: Electric, Grass, Ground, Water
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS || move.type == TYPES::GROUND || move.type == TYPES::WATER) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::GROUND:
            //Weak: WATER, GRASS, ICE
            if (move.type == TYPES::WATER || move.type == TYPES::GRASS || move.type == TYPES::ICE) {
                modifier = modifier * 2;
            }
            //Resist: POISON, ROCK
            if (move.type == TYPES::POISON || move.type == TYPES::ROCK) {
                modifier = modifier / 2;
            }
            //Immune: ELECTRIC
            if (move.type == TYPES::ELECTRIC) {
                modifier = 0;
            }
            break;

        case TYPES::ICE:
            //Weak: FIGHTING, ROCK, STEEL, FIRE
            if (move.type == TYPES::FIGHTING || move.type == TYPES::ROCK || move.type == TYPES::STEEL || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: ICE
            if (move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::NORMAL:
            //Weak: FIGHTING
            if (move.type == TYPES::FIGHTING) {
                modifier = modifier * 2;
            }
            //Immune: GHOST
            if (move.type == TYPES::GHOST) {
                modifier = 0;
            }
            break;

        case TYPES::POISON:
            //Weak: Ground, Psychic
            if (move.type == TYPES::GROUND || move.type == TYPES::PSYCHIC) {
                modifier = modifier * 2;
            }
            //Resist: Fighting, Poison, Bug, Grass, FAIRY
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG ||
                move.type == TYPES::GRASS || move.type == TYPES::FAIRY) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::PSYCHIC:
            //Weak: BUG, DARK, GHOST
            if (move.type == TYPES::FIGHTING || move.type == TYPES::POISON || move.type == TYPES::BUG) {
                modifier = modifier * 2;
            }
            //Resist: FIGHTING, PSYCHIC
            if (move.type == TYPES::FIGHTING || move.type == TYPES::PSYCHIC) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::ROCK:
            //Weak: FIGHTING, GROUND, STEEL, WATER, GRASS
            if (move.type == TYPES::FIGHTING || move.type == TYPES::GROUND || move.type == TYPES::STEEL ||
                move.type == TYPES::GRASS || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, POISON, FIRE
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::POISON || move.type == TYPES::FIRE) {
                modifier = modifier / 2;
            }
            break;

        case TYPES::STEEL:
            //Weak: GROUND, FIGHTING, FIRE
            if (move.type == TYPES::GROUND || move.type == TYPES::FIGHTING || move.type == TYPES::FIRE) {
                modifier = modifier * 2;
            }
            //Resist: NORMAL, FLYING, ROCK, BUG, STEEL, GRASS, PSYCHIC, ICE, FAIRY, DRAGON
            if (move.type == TYPES::NORMAL || move.type == TYPES::FLYING || move.type == TYPES::ROCK || move.type == TYPES::BUG ||
                move.type == TYPES::STEEL || move.type == TYPES::GRASS || move.type == TYPES::PSYCHIC || move.type == TYPES::ICE ||
                move.type == TYPES::FAIRY || move.type == TYPES::DRAGON) {
                modifier = modifier / 2;
            }
            //Immune: POISON
            if (move.type == TYPES::POISON) {
                modifier = 0;
            }
            break;

        case TYPES::WATER:
            //Weak: Electric, Grass
            if (move.type == TYPES::ELECTRIC || move.type == TYPES::GRASS) {
                modifier = modifier * 2;
            }
            //Resist: Steel, Fire, Water, Ice
            if (move.type == TYPES::STEEL || move.type == TYPES::FIRE || move.type == TYPES::WATER || move.type == TYPES::ICE) {
                modifier = modifier / 2;
            }
            break;
        }
    }

    if (modifier > 1) {
        superEff++;
    }
    else if (modifier == 0) {
        noEff++;
    }
    else if (modifier < 1) {
        notVeryEff++;
    }

    damage = (int)(damage * modifier);
    dmgArray[0] = damage;
    dmgArray[1] = superEff;
    dmgArray[2] = notVeryEff;
    dmgArray[3] = noEff;  // Update as necessary for no effect cases
}

__device__ void damageCalcGPU(Pokemon& attacker, Pokemon& defender, Move move, int seed) {
    // Variables
    double critCalc = 0;
    double stab = 1.0;
    double randFact = 0;
    double force = 0;
    double object = 0;

    // Physical or special attack calculation
    if (move.category == 1) {
        force = attacker.statAtk[1];
        object = defender.statDef[1];
    }
    else if (move.category == 2) {
        force = attacker.statSpa[1];
        object = defender.statSpd[1];
    }

    // Critical hit calculation using random number generator
    int randVal = getRandom(seed, 0); // You can pass a custom seed and offset
    if (randVal < (attacker.statSpe[1] / 2)) {
        critCalc = 1.5;
    }
    else {
        critCalc = 1.0;
    }

    // Random factor calculation
    randFact = (randVal + 85) / 100.0;

    // Same Type Attack Bonus (STAB)
    if (move.type == attacker.type1 || move.type == attacker.type2) {
        stab = 1.5;
    }

    // Base damage calculation
    int baseDamage = static_cast<int>((((22) * move.power * (force / object)) / (50.0 + 2)) * critCalc * randFact * stab);

    // Get type multiplier
    typeMultiplierGPU(baseDamage, move, defender);
    baseDamage = dmgArray[0];

    // Apply damage to defender's HP
    defender.healthPoints -= baseDamage;
}

__device__ void statusCalcGPU(Pokemon& attacker, Pokemon& defender, Move move) {
    // {buff/debuff, atk, def, spa, spd, spe, acc, eva}
    int* effect = move.statEffect;
    double mult;
    int multStage;

    if (effect[0] == 0) { // If buffing self
        if (effect[1] != 0) { // Buff attack
            multStage = effect[1];
            attacker.statAtk[0] += multStage;
            if (attacker.statAtk[0] > 6) {
                attacker.statAtk[0] = 6;
            }
            else if (attacker.statAtk[0] < -6) {
                attacker.statAtk[0] = -6;
            }
            multStage = attacker.statAtk[0];
            mult = statMultiplierGPU[multStage];
            int newAtk = static_cast<int>(mult * attacker.statAtk[1]);
            attacker.statAtk[1] = newAtk;
            /*std::cout << attacker.getPokeName() << "'s attack rose!" << std::endl;*/
        }
        if (effect[2] != 0) { // Buff defense
            multStage = effect[2];
            attacker.statDef[0] += multStage;
            if (attacker.statDef[0] > 6) {
                attacker.statDef[0] = 6;
            }
            else if (attacker.statDef[0] < -6) {
                attacker.statDef[0] = -6;
            }
            multStage = attacker.statDef[0];
            mult = statMultiplierGPU[multStage];
            int newDef = static_cast<int>(mult * attacker.statDef[1]);
            attacker.statDef[1] = newDef;
            //std::cout << attacker.getPokeName() << "'s defense rose!" << std::endl;
        }
        if (effect[3] != 0) { // Buff special attack
            multStage = effect[3];
            attacker.statSpa[0] += multStage;
            if (attacker.statSpa[0] > 6) {
                attacker.statSpa[0] = 6;
            }
            else if (attacker.statSpa[0] < -6) {
                attacker.statSpa[0] = -6;
            }
            multStage = attacker.statSpa[0];
            mult = statMultiplierGPU[multStage];
            int newSpa = static_cast<int>(mult * attacker.statSpa[1]);
            attacker.statSpa[1] = newSpa;
            //std::cout << attacker.getPokeName() << "'s special attack rose!" << std::endl;
        }
        if (effect[4] != 0) { // Buff special defense
            multStage = effect[4];
            attacker.statSpd[0] += multStage;
            if (attacker.statSpd[0] > 6) {
                attacker.statSpd[0] = 6;
            }
            else if (attacker.statSpd[0] < -6) {
                attacker.statSpd[0] = -6;
            }
            multStage = attacker.statSpd[0];
            mult = statMultiplierGPU[multStage];
            int newSpd = static_cast<int>(mult * attacker.statSpd[1]);
            attacker.statSpd[1] = newSpd;
            //std::cout << attacker.getPokeName() << "'s special defense rose!" << std::endl;
        }
        if (effect[5] != 0) { // Buff speed
            multStage = effect[5];
            attacker.statSpe[0] += multStage;
            if (attacker.statSpe[0] > 6) {
                attacker.statSpe[0] = 6;
            }
            else if (attacker.statSpe[0] < -6) {
                attacker.statSpe[0] = -6;
            }
            multStage = attacker.statSpe[0];
            mult = statMultiplierGPU[multStage];
            int newSpe = static_cast<int>(mult * attacker.statSpe[1]);
            attacker.statSpe[1] = newSpe;
            //std::cout << attacker.getPokeName() << "'s speed rose!" << std::endl;
        }
        if (effect[6] != 0) { // Buff accuracy
            //std::cout << attacker.getPokeName() << "'s accuracy rose!" << std::endl;
        }
        if (effect[7] != 0) { // Buff evasion
            //std::cout << attacker.getPokeName() << "'s evasion rose!" << std::endl;
        }
    }
    else if (effect[0] == 1) { // Debuffing opponent
        if (effect[1] != 0) { // Debuff attack
            multStage = effect[1];
            defender.statAtk[0] += multStage;
            if (defender.statAtk[0] > 6) {
                defender.statAtk[0] = 6;
            }
            else if (defender.statAtk[0] < -6) {
                defender.statAtk[0] = -6;
            }
            multStage = defender.statAtk[0];
            mult = statMultiplierGPU[multStage];
            int newAtk = static_cast<int>(mult * defender.statAtk[1]);
            defender.statAtk[1] = newAtk;
            //std::cout << defender.getPokeName() << "'s attack fell!" << std::endl;
        }
        if (effect[2] != 0) { // Debuff defense
            multStage = effect[2];
            defender.statDef[0] += multStage;
            if (defender.statDef[0] > 6) {
                defender.statDef[0] = 6;
            }
            else if (defender.statDef[0] < -6) {
                defender.statDef[0] = -6;
            }
            multStage = defender.statDef[0];
            mult = statMultiplierGPU[multStage];
            int newDef = static_cast<int>(mult * defender.statDef[1]);
            defender.statDef[1] = newDef;
            //std::cout << defender.getPokeName() << "'s defense fell!" << std::endl;
        }
        if (effect[3] != 0) { // Debuff special attack
            multStage = effect[3];
            defender.statSpa[0] += multStage;
            if (defender.statSpa[0] > 6) {
                defender.statSpa[0] = 6;
            }
            else if (defender.statSpa[0] < -6) {
                defender.statSpa[0] = -6;
            }
            multStage = defender.statSpa[0];
            mult = statMultiplierGPU[multStage];
            int newSpa = static_cast<int>(mult * defender.statSpa[1]);
            defender.statSpa[1] = newSpa;
            //std::cout << defender.getPokeName() << "'s special attack fell!" << std::endl;
        }
        if (effect[4] != 0) { // Debuff special defense
            multStage = effect[4];
            defender.statSpd[0] += multStage;
            if (defender.statSpd[0] > 6) {
                defender.statSpd[0] = 6;
            }
            else if (defender.statSpd[0] < -6) {
                defender.statSpd[0] = -6;
            }
            multStage = defender.statSpd[0];
            mult = statMultiplierGPU[multStage];
            int newSpd = static_cast<int>(mult * defender.statSpd[1]);
            defender.statSpd[1] = newSpd;
            //std::cout << defender.getPokeName() << "'s special defense fell!" << std::endl;
        }
        if (effect[5] != 0) { // Debuff speed
            multStage = effect[5];
            defender.statSpe[0] += multStage;
            if (defender.statSpe[0] > 6) {
                defender.statSpe[0] = 6;
            }
            else if (defender.statSpe[0] < -6) {
                defender.statSpe[0] = -6;
            }
            multStage = defender.statSpe[0];
            mult = statMultiplierGPU[multStage];
            int newSpe = static_cast<int>(mult * defender.statSpe[1]);
            defender.statSpe[1] = newSpe;
            //std::cout << defender.getPokeName() << "'s speed fell!" << std::endl;
        }
        if (effect[6] != 0) { // Debuff accuracy
            //std::cout << defender.getPokeName() << "'s accuracy fell!" << std::endl;
        }
        if (effect[7] != 0) { // Debuff evasion
            //std::cout << defender.getPokeName() << "'s evasion fell!" << std::endl;
        }
    }
}

__device__ void battleGPU(Pokemon& pokemon1, Pokemon& pokemon2, Pokemon& winner, int seed) {
    int moveIndex;
    Move selected = pokemon1.moves[0]; // initial move bc I don't want to figure out the actual problem

    while (pokemon1.healthPoints >= 1 && pokemon2.healthPoints >= 1) {
        if (pokemon1.statSpe[1] > pokemon2.statSpe[1]) {
            moveIndex = getRandom(seed, 1) % sizeof(pokemon1.moves) / sizeof(pokemon1.moves[0]);
            selected = pokemon1.moves[moveIndex];

            if (selected.category == 3) {
                statusCalcGPU(pokemon1, pokemon2, selected);
            }
            else {
                damageCalcGPU(pokemon1, pokemon2, selected, seed);
            }

            if (pokemon2.healthPoints < 1) break;

            moveIndex = getRandom(seed, 2) % sizeof(pokemon2.moves) / sizeof(pokemon2.moves[0]);
            selected = pokemon2.moves[moveIndex];

            if (selected.category == 3) {
                statusCalcGPU(pokemon2, pokemon1, selected);
            }
            else {
                damageCalcGPU(pokemon2, pokemon1, selected, seed);
            }
        }
        else {
            moveIndex = getRandom(seed, 3) % sizeof(pokemon2.moves) / sizeof(pokemon2.moves[0]);
            selected = pokemon2.moves[moveIndex];

            if (selected.category == 3) {
                statusCalcGPU(pokemon2, pokemon1, selected);
            }
            else {
                damageCalcGPU(pokemon2, pokemon1, selected, seed);
            }

            if (pokemon1.healthPoints < 1) break;

            moveIndex = getRandom(seed, 4) % sizeof(pokemon1.moves) / sizeof(pokemon1.moves[0]);
            selected = pokemon1.moves[moveIndex];

            if (selected.category == 3) {
                statusCalcGPU(pokemon1, pokemon2, selected);
            }
            else {
                damageCalcGPU(pokemon1, pokemon2, selected, seed);
            }
        }
    }

    if (pokemon2.healthPoints < 1) {
        winner = pokemon1;
    }
    else {
        winner = pokemon2;
    }
}


// Kernel to simulate many battles
__global__ void battleKernel(Pokemon* p1Array, Pokemon* p2Array, Pokemon* results, int numBattles) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numBattles) {
        battleGPU(p1Array[idx], p2Array[idx], results[idx], idx + 1234); // Unique seed per thread
    }
}

bool pokeBattleGPU(Pokemon* pokemon1, Pokemon* pokemon2, Pokemon* results, int NUM_POKEMON) {
    hipError_t status;
    Pokemon* d_p1, * d_p2;
    Pokemon* r1;

    // Allocate memory on the device
    hipMalloc((void**)&d_p1, sizeof(Pokemon) * NUM_POKEMON);
    hipMalloc((void**)&d_p2, sizeof(Pokemon) * NUM_POKEMON);
    hipMalloc((void**)&r1, sizeof(Pokemon) * NUM_POKEMON);

    // Copy input Pok�mon from host to device
    hipMemcpy(d_p1, &pokemon1, sizeof(Pokemon), hipMemcpyHostToDevice);
    hipMemcpy(d_p2, &pokemon2, sizeof(Pokemon), hipMemcpyHostToDevice);

    dim3 blockSize(TILE_SIZE);
    dim3 gridSize((NUM_POKEMON + TILE_SIZE - 1) / TILE_SIZE);

    // Launch 1 thread to simulate 1 battle
    battleKernel << <gridSize, blockSize >> > (d_p1, d_p2, results, NUM_POKEMON);
    hipDeviceSynchronize(); // Wait for GPU to finish

    status = hipGetLastError();
    if (status != hipSuccess) {
        std::cerr << "Kernel failed: " << hipGetErrorString(status) << std::endl;
        hipFree(d_p1);
        hipFree(d_p2);
        hipFree(r1);
        return false;
    }

    hipMemcpy(&results, r1, sizeof(Pokemon), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_p1);
    hipFree(d_p2);
    hipFree(r1);

    return true;
}